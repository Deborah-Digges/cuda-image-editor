#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include "SobelFilter.h"
#include <cmath>

/* Declaration of kernel functions.
 */
__global__ void sobelFilter(const uchar4* d_inputImageRGBA , uchar4* d_outputImageRGBA , const float* d_filterX , const float* d_filterY , int numRows , int numCols , int filterWidth);
__device__ unsigned char gray(const uchar4 & pix);
__device__ char threshold(char input);


SobelFilter * SobelFilter::instance = NULL;


SobelFilter::SobelFilter()
{
}

/* The client can create an object of SobelFilter class only through the static factory function.
 * It returns a pointer to the only currently existing instance of the class.
 */
SobelFilter * SobelFilter::factory()
{
	if(SobelFilter::instance == NULL)
	{
		SobelFilter::instance = new SobelFilter();
	}
	return SobelFilter::instance;
	
}

/* The input image is converted to the appropriate format for processing.
 * Helper methods are called for allocating memory on the GPU and copying from the CPU to GPU , making the filter and performing the blurring
 * The output image is copied from GPU to CPU memory
 * The output image is converted to cv::Mat format and returned to the client.
 */
cv::Mat SobelFilter::operator()(const cv::Mat & image)
{
	std::cout << "functor\n";
	cv::cvtColor(image , imageInputRGBA , CV_BGR2RGBA);
	numRows = imageInputRGBA.rows;
	numCols = imageInputRGBA.cols;
	imageOutputRGBA.create(image.rows, image.cols, CV_8UC4);
	allocateMemory();
	makeFilter();
	wrapperFilter();
	hipMemcpy(h_outputImageRGBA, d_outputImageRGBA , sizeof(uchar4) * numRows * numCols, hipMemcpyDeviceToHost);
	cv::Mat output(imageOutputRGBA.rows, imageOutputRGBA.cols, CV_8UC4, (void*)h_outputImageRGBA);
	resetFilter();							//CV_8UC4
	return output;

}

/* Allocates memory on the GPU for the structures needed and initializes them to 0.
 * Copies the source image from the CPU to the GPU memory.
 */

void SobelFilter::allocateMemory()
{
	std::cout << "allocate\n";
	const size_t numPixels = numRows * numCols;
	h_inputImageRGBA  = (uchar4 *)imageInputRGBA.ptr<unsigned char>(0);
	h_outputImageRGBA = (uchar4 *)imageOutputRGBA.ptr<unsigned char>(0);
	hipMalloc(&d_inputImageRGBA, sizeof(uchar4) * numPixels);
	hipMalloc(&d_outputImageRGBA, sizeof(uchar4) * numPixels);
	hipMemset(d_outputImageRGBA, 0, numPixels * sizeof(uchar4));
	hipMemcpy(d_inputImageRGBA, h_inputImageRGBA, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice);
	
}
/* Creates two 3x3 filters for differentiation of the image intensity function in both the x and the y direction.
 * The 2 filters are created in host memory and then copied to GPU memory.
 */


void SobelFilter::makeFilter()
{
	std::cout << "make\n";
	filterWidth = 3;
	int index = 0;
	int x[] = {-1 , 0 , 1 , -2 , 0 , 2 , -1 , 0 , 1};
	int y[] = {1 , 2 , 1 , 0 , 0 , 0 , -1 , -2 , -1};
	//int y[] = {3 , 10 , 3 , 0 , 0 , 0 , -3 , -10 , -3};
	//int x[] = {3 , 0  , -3 , 10 , 0 , -10 , 3 , 0 , -3};
	//create and fill the filter we will convolve with
	h_filterX = new float[filterWidth * filterWidth];
	h_filterY = new float[filterWidth * filterWidth];
	
	for(index = 0; index < filterWidth * filterWidth; ++index)
	{
		h_filterX[index] = x[index];
		h_filterY[index] = y[index];	
	}	
		
	
	hipMalloc(&d_filterX , filterWidth * filterWidth * sizeof(float));
	hipMalloc(&d_filterY , filterWidth * filterWidth * sizeof(float));
	hipMemcpy(d_filterX , h_filterX , filterWidth * filterWidth * sizeof(float) , hipMemcpyHostToDevice);
	hipMemcpy(d_filterY , h_filterY , filterWidth * filterWidth * sizeof(float) , hipMemcpyHostToDevice);
}

/* Wrapper to call the sobel kernel.
 * Initializes the block and grid dimensions and calls the sobel filter kernel 
*/
void SobelFilter::wrapperFilter()
{
	std::cout << "reset\n";
	const int BLOCK_WIDTH =  32;
	const dim3 blockSize(BLOCK_WIDTH , BLOCK_WIDTH);
	const dim3 gridSize((numCols/BLOCK_WIDTH) + 1 , (numRows/BLOCK_WIDTH) + 1 );
	
	sobelFilter<<<gridSize,blockSize>>>(d_inputImageRGBA , d_outputImageRGBA , d_filterX , d_filterY , numRows , numCols , filterWidth);
        hipDeviceSynchronize(); hipGetLastError();
  	
}

/* Helper function for returning a grey scale pixel for a given colored pixel */
__device__ unsigned char gray(const uchar4 & pix)
{
	return (0.3f * pix.x + 0.59f * pix.y + 0.11f * pix.z);
}

/* Kernel that performs the edge detection using sobel filter.
 * It first applies by convolution the filter for differentiation in the x direction and then the filter for diff. in the y direction.
 * The magnitude of the differential of the image intensity function is found by pythogoras theorem.
 * The magnitude is compared to a threshold value; If it is greater than a threshold value, the pixel is made black , else the pixel is made 
 * white.
 */
__global__ void sobelFilter(const uchar4* d_inputImageRGBA , uchar4* d_outputImageRGBA , const float* d_filterX , const float* d_filterY , int numRows , int numCols , int filterWidth)
{
	const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x , blockIdx.y * blockDim.y + threadIdx.y);
	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
	
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
        {
		return;
	}
		
	float sumX = 0.f;
	float sumY = 0.f;
	int row = 0;
	int col = 0;
	float g = 0.f;
	float filterX_value =  0.f;
	float filterY_value = 0.f;
	int halfWidth = filterWidth/2;
	
	for(row = -1 ; row <= 1 ; ++row)
	{
		for(col = -1 ; col <= 1 ; ++col)
		{
			int image_r = min(max(thread_2D_pos.y + row, 0), (numRows - 1));
			int image_c = min(max(thread_2D_pos.x + col, 0), (numCols - 1));
			g = gray(d_inputImageRGBA[(image_r) * numCols + (image_c)]);
			filterX_value = d_filterX[(row + halfWidth) * filterWidth + (col + halfWidth)];
			filterY_value = d_filterY[(row + halfWidth) * filterWidth + (col + halfWidth)];
			sumX += g * filterX_value;
			sumY += g * filterY_value;
                        
		}
	}
	//unsigned char p = abs((sumX/8.0)) + abs((sumY/8.0));
	unsigned char p = sqrt( abs((sumX/8.0)) * abs((sumX/8.0)) + abs((sumY/8.0)) * abs((sumY/8.0)));
	p = threshold(p);
	uchar4 output;
	output.x = 0;
	output.y = 0;
	output.z = 0;
	output.w = p;
	
	d_outputImageRGBA[thread_1D_pos] = output;//make_uchar4(0, 255 , 255 , 255);	
	
	
}

__device__ char threshold(char input)
{
	if ( input >= 8 )
	{
		return (char )255;
	}
	else
	{
		return (char )0 ;
	}
} 

/* After each image has been processed, the SobelFilter object is reset so that the data structures for each image are created according 
 * the size of the input image.
 * All instance members are taken care of here.
 * The destructor here is therefore a dummy destructor.
 */
void SobelFilter::resetFilter()
{
	std::cout <<"reset\n";
	hipFree(d_inputImageRGBA);
	hipFree(d_outputImageRGBA);
	hipFree(d_filterX);
	hipFree(d_filterY);
	delete [] h_filterX;
	delete [] h_filterY;
	h_inputImageRGBA = NULL;
	h_outputImageRGBA = NULL;
	d_inputImageRGBA = NULL;
	d_outputImageRGBA = NULL;
	h_filterX = NULL;
	h_filterY = NULL;
	d_filterX = NULL;
	d_filterY = NULL;
	filterWidth = 0;
	numRows = 0;
	numCols = 0;
}




